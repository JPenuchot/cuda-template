#include <iostream>
#include <numeric>
#include <vector>

#include <hip/hip_runtime.h>

using namespace std;

template<typename T>
__global__ void vadd(T* a, T* b, T* dest)
{
  dest[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

template<typename T>
void cu_add(T* a, T* b, T* dest, size_t size)
{
  vadd<<<1, size>>>(a, b, dest);
}

int main()
{
  vector<float> va(100);
  vector<float> vb(100);

  fill(va.begin(), va.end(), 0.f);
  iota(vb.begin(), vb.end(), 0.f);

  float* dev_a = nullptr;
  float* dev_b = nullptr;

  hipMalloc(&dev_a, va.size() * sizeof(float));
  hipMalloc(&dev_b, vb.size() * sizeof(float));

  hipMemcpy( dev_a, va.data()
            , va.size() * sizeof(float)
            , hipMemcpyHostToDevice);

  hipMemcpy( dev_b, vb.data()
            , vb.size() * sizeof(float)
            , hipMemcpyHostToDevice);


  cu_add(dev_a, dev_b, dev_a, va.size());

  hipMemcpy( va.data(), dev_a
            , va.size() * sizeof(float), hipMemcpyDeviceToHost);

  for(auto& e : va) cout << e << '\n';

  return 0;
}
