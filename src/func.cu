#include "hip/hip_runtime.h"
#include "func.hpp"

__global__ void ke_add(float* a, float* b, float* dest)
{
  dest[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

__host__ void cu_add(float* a, float* b, float* dest, size_t size)
{
  ke_add <<<1, size>>> (a, b, dest);
}
