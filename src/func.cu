#include "hip/hip_runtime.h"
#include "func.hpp"

__global__ void vadd(float* a, float* b, float* dest)
{
  dest[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void cu_add(float* a, float* b, float* dest, size_t size)
{
  vadd<<<1, size>>>(a, b, dest);
}
